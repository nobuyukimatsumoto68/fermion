#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cassert>
#include <fstream>

// #include <omp.h>

// #include <Eigen/Dense>
// #include <Eigen/Eigenvalues>

#include "typedefs_cuda.hpp"
#include "constants.hpp"
#include "header_cuda.hpp"

// ======================================



// int main(){
int main(int argc, char **argv){

  if (argc>1){
    nu = atoi(argv[1]);
    // printf("%s\n", argv[i]);
  }
  const std::string description = "Lx"+std::to_string(Lx)+"Ly"+std::to_string(Ly)+"nu"+std::to_string(nu);
  // description = "Lx"+std::to_string(Lx)+"Ly"+std::to_string(Ly)+"nu"+std::to_string(nu);

  int device_num;
  cudacheck(hipGetDeviceCount(&device_num));
  hipDeviceProp_t device_prop[device_num];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "dev = " << device_prop[0].name << std::endl;
  cudacheck(hipSetDevice(0));// "TITAN V"
  std::cout << "(GPU device is set.)" << std::endl;


  // -----

  Complex *e, *Dinv;
  e = (Complex*)malloc(N*CD);
  Dinv = (Complex*)malloc(N*CD);

  {
    int xx = 0, yy = 0;

    set2zero(e, N);
    e[ 2*idx(xx, yy) ] = cplx(1.0);
    multDdagger_wrapper( e, e);

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_0_0_0_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }

    set2zero(e, N);
    e[ 2*idx(xx, yy)+1] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_0_0_1_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }
  }

  //------------------------------------------

  {
    int xx = -1, yy = 0;

    set2zero(e, N);
    e[ 2*idx(xx, yy) ] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_m1_0_0_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }

    set2zero(e, N);
    e[ 2*idx(xx, yy)+1] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_m1_0_1_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }
  }

  //------------------------------------------

  {
    int xx = 1, yy = -1;

    set2zero(e, N);
    e[ 2*idx(xx, yy) ] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_1_m1_0_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }

    set2zero(e, N);
    e[ 2*idx(xx, yy)+1] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_1_m1_1_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }
  }

  //------------------------------------------

  {
    int xx = 0, yy = 1;

    set2zero(e, N);
    e[ 2*idx(xx, yy) ] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_0_1_0_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }

    set2zero(e, N);
    e[ 2*idx(xx, yy)+1] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_0_1_1_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }
  }

  //

  {
    int xx = Lx/3, yy = Lx/3;

    set2zero(e, N);
    e[ 2*idx(xx, yy) ] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_T_T_0_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }

    set2zero(e, N);
    e[ 2*idx(xx, yy)+1] = cplx(1.0);
    multDdagger_wrapper( e, e );

    set2zero(Dinv, N);
    solve(Dinv, e);

    {
      std::ofstream of( dir_data+description+"Dinv_T_T_1_cuda.dat",
                        std::ios::out | std::ios::binary | std::ios::trunc);
      if(!of) assert(false);

      double tmp = 0.0;
      for(Idx i=0; i<N; i++){
        tmp = real(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );

        tmp = imag(Dinv[i]);
        of.write((char*) &tmp, sizeof(double) );
      }
    }
  }

  //

  free( e );
  free( Dinv );
  hipDeviceReset();

  return 0;
}

