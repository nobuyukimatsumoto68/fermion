#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cassert>
#include <cstring>
#include <vector>
#include <fstream>
#include <algorithm>

#include "typedefs_cuda.hpp"
#include "constants.hpp"
#include "header_cuda.hpp"
#include "header_cusolver.hpp"


// using Idx = long long int;
// using Idx = size_t;


int compare (const void * a, const void * b)
{
  return ( *(Idx*)a - *(Idx*)b );
}

int main(int argc, char **argv){
  std::cout << std::scientific << std::setprecision(15) << std::endl;

  int nu=3;

  if (argc>1){
    for (int i = 0; i < argc; i++) {
      nu = atoi(argv[1]);
      printf("%s\n", argv[i]);
    }
  }
  const std::string description = "Lx"+std::to_string(Lx)+"Ly"+std::to_string(Ly)+"nu"+std::to_string(nu);

  int device;
  cudacheck(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "dev = " << device_prop[0].name << std::endl;
  cudacheck(hipSetDevice(0));// "TITAN V"
  std::cout << "(GPU device is set.)" << std::endl;


  // -----

  std::cout << "calculating D" << std::endl;

  const Idx N2 = N*N;
  const Idx Neff = N*2/3;

  Complex *D_removed, *LU;

  {
    Complex *e, *D;

    // =====

    std::cout << "-- memory alloc" << std::endl;
    D = (Complex*)malloc(N2*CD);
    set2zero(D, N*N);
    e = (Complex*)malloc(N*CD);

    std::cout << "-- setting" << std::endl;
    for(Idx i=0; i<N; i++){
      set2zero(e, N);
      e[i] = cplx(1.0);
      multD_wrapper( D+i*N, e, nu ); // column major
    }

    free( e );

    //=====

    D_removed = (Complex*)malloc(Neff*Neff*CD);
    set2zero(D_removed, Neff*Neff);

    {
      // std::vector<Idx> vacant;
      // Idx vacant[N/3];
      Idx *vacant;

      vacant = (Idx*)malloc(N/3*sizeof(Idx));
      set2zero(vacant, N/3);

      Idx ii=0;
      for(Idx x=0; x<Lx; x++){
        for(Idx y=0; y<Ly; y++){
          const Idx idx1 = 2*idx(x,y);
          if( !is_site(x,y) ) {
            // vacant.push_back( idx1 );
            // vacant.push_back( idx1+1 );
            vacant[ii]=idx1;
            ii++;
            vacant[ii]=idx1+1;
            ii++;
          }
        }}
      // std::sort(vacant.begin(),vacant.end());
      // std::sort(std::begin(vacant), std::end(vacant));
      std::cout << "sorting..." << std::endl;
      qsort( vacant, N/3, sizeof(Idx), compare );

      Idx idx_tot = 0;

      Idx js=0;
      for(Idx j=0; j<N; j++){
        if(j==vacant[js]){
          js++;
          continue;
        }
        Idx is = 0;
        for(Idx i=0; i<N; i++){
          if(i==vacant[is]){
            is++;
            continue;
          }

          D_removed[idx_tot] = D[i*N+j];
          idx_tot++;
        }}

      free( vacant );
    }

    free( D );
  }

  //=======

  std::cout << "starting LU" << std::endl;

  LU = (Complex*)malloc(Neff*Neff*CD);
  set2zero(LU, Neff*Neff);

  {
    // https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/Xgetrf/cusolver_Xgetrf_example.cu
    // https://github.com/mnicely/cusolver_examples/blob/main/lu_decomposition_cusolver.cu


    // =========================================
    // cusolver
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int64_t m = Neff;
    const int64_t lda = m;

    int info = 0;

    Complex *d_A;
    int64_t *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr;  /* error info */

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr;              /* device workspace for getrf */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void *h_work = nullptr;              /* host workspace for getrf */


    /* step 1: create cusolver handle, bind a stream */
    cudacheck( hipsolverDnCreate( &cusolverH ) );
    cudacheck( hipStreamCreate( &stream ) );
    cudacheck( hipsolverSetStream( cusolverH, stream ) );


    /* Create advanced params */
    hipsolverDnParams_t params;
    cudacheck( hipsolverDnCreateParams( &params ) );



    /* step 2: copy A to device */
    std::cout << "Memory allocation" << std::endl;
    cudacheck( hipMalloc( &d_A, Neff*Neff*CD) );
    cudacheck( hipMalloc( &d_Ipiv, sizeof( int64_t ) * Neff ) );
    cudacheck( hipMalloc( &d_info, sizeof( int ) ) );

    cudacheck(hipMemcpy(d_A, D_removed, Neff*Neff*CD, H2D));


    /* step 3: query working space of getrf */
    cudacheck( hipsolverDnXgetrf_bufferSize( cusolverH, NULL, Neff, Neff,
                                            HIP_C_64F, d_A, Neff, HIP_C_64F,
                                            &workspaceInBytesOnDevice, &workspaceInBytesOnHost ) );


    cudacheck(hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice));

    if (0 < workspaceInBytesOnHost) {
      h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
      if (h_work == nullptr) {
        throw std::runtime_error("Error: h_work not allocated.");
      }
    }


    /* step 4: LU factorization */
    std::cout << "Executing LU" << std::endl;

    cudacheck(hipsolverDnXgetrf(cusolverH, params, m, m, HIP_C_64F,
                               d_A, lda, d_Ipiv, HIP_C_64F, d_work,
                               workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info));

    cudacheck(hipMemcpy(&info, d_info, sizeof(int), D2H));
    cudacheck(hipMemcpy(LU, d_A, Neff*Neff*CD, D2H));

    std::printf("after Xgetrf: info = %d\n", info);
    if (0 > info) {
      std::printf("%d-th parameter is wrong \n", -info);
      exit(1);
    }

    std::vector<double> zeros;
    double log_abs_det_nozero = 0.0;
    // std::complex<double> det = 1.0;
    for(Idx i=0; i<Neff; i++) {
      // assert( imag(LU[i*Neff+i])<1.0e-14 );
      // log_det += std::log( real(LU[i*Neff+i]) );
      std::complex<double> tmp = real(LU[i*Neff+i]) + std::complex<double>(0.0,1.0)*imag(LU[i*Neff+i]);
      if( abs(arg(tmp))>1.0e-13 ) std::cout << "arg>0: " << tmp << std::endl;
      if( abs(tmp)>1.0e-13 ) log_abs_det_nozero += std::log( abs(tmp) );
      else zeros.push_back( std::abs(tmp) );
    }
    std::cout << "nu = " << nu << std::endl
      //<< "log det = " << log_det << std::endl;
              << "log_abs_det_nozero = " << log_abs_det_nozero << std::endl;

    std::cout << "zeros: " << std::endl;
    for(auto elem : zeros) std::cout << elem << " ";
    std::cout << std::endl;


    // ===========================================

    cudacheck( hipFree( d_A ) );
    cudacheck( hipFree( d_Ipiv ) );
    cudacheck( hipFree( d_info ) );
    cudacheck( hipFree( d_work ) );
    free(h_work);

    cudacheck(hipsolverDnDestroyParams(params));
    cudacheck( hipsolverDnDestroy( cusolverH ) );
    cudacheck( hipStreamDestroy( stream ) );
  }


  free( D_removed );
  free( LU );
  hipDeviceReset();

  return 0;
}

