#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cassert>
#include <cstring>
#include <fstream>
#include <algorithm>

#include "typedefs_cuda.hpp"
#include "constants.hpp"
#include "header_cuda.hpp"
#include "header_cusolver.hpp"

// ======================================

//  #define FMULS_GETRF(m_, n_) ( ((m_) < (n_)) \
//      ? (0.5 * (m_) * ((m_) * ((n_) - (1./3.) * (m_) - 1. ) + (n_)) + (2. / 3.) * (m_)) \
//      : (0.5 * (n_) * ((n_) * ((m_) - (1./3.) * (n_) - 1. ) + (m_)) + (2. / 3.) * (n_)) )
//  #define FADDS_GETRF(m_, n_) ( ((m_) < (n_)) \
//      ? (0.5 * (m_) * ((m_) * ((n_) - (1./3.) * (m_)      ) - (n_)) + (1. / 6.) * (m_)) \
//      : (0.5 * (n_) * ((n_) * ((m_) - (1./3.) * (n_)      ) - (m_)) + (1. / 6.) * (n_)) )
// #define FLOPS_ZGETRF(m_, n_) (6. * FMULS_GETRF((double)(m_), (double)(n_)) + 2.0 * FADDS_GETRF((double)(m_), (double)(n_)) )


using Idx = long int;


int main(){
  std::cout << std::scientific << std::setprecision(15) << std::endl;

  int device;
  cudacheck(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "dev = " << device_prop[0].name << std::endl;
  cudacheck(hipSetDevice(0));// "TITAN V"
  std::cout << "(GPU device is set.)" << std::endl;


  // -----

  Complex *e, *D, *D_removed, *LU;
  e = (Complex*)malloc(N*CD);

  const Idx N2 = N*N;
  const Idx Neff = N*2/3;

  D = (Complex*)malloc(N2*CD);
  set2zero(D, N*N);

  D_removed = (Complex*)malloc(Neff*Neff*CD);
  set2zero(D_removed, Neff*Neff);
  // Complex D_removed[Neff*Neff];

  LU = (Complex*)malloc(Neff*Neff*CD);
  set2zero(LU, Neff*Neff);

  for(Idx i=0; i<N; i++){
    set2zero(e, N);
    e[i] = cplx(1.0);
    multD_wrapper( D+i*N, e ); // column major
  }


  {
    std::vector<int> vacant;
    for(int x=0; x<Lx; x++){
      for(int y=0; y<Ly; y++){
        const Idx idx1 = 2*idx(x,y);
        if( !is_site(x,y) ) {
          vacant.push_back( idx1 );
          vacant.push_back( idx1+1 );
        }
      }}
    std::sort(vacant.begin(),vacant.end());

    int idx_tot = 0;

    int js=0;
    for(int j=0; j<N; j++){
      if(j==vacant[js]){
        js++;
        continue;
      }
      int is = 0;
      for(int i=0; i<N; i++){
        if(i==vacant[is]){
          is++;
          continue;
        }

        D_removed[idx_tot] = D[i*N+j];
        idx_tot++;
      }}
  }


  {
    // https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/Xgetrf/cusolver_Xgetrf_example.cu
    // https://github.com/mnicely/cusolver_examples/blob/main/lu_decomposition_cusolver.cu


    // =========================================
    // cusolver
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int64_t m = Neff;
    const int64_t lda = m;

    int info = 0;

    Complex *d_A;
    int64_t *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr;  /* error info */

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr;              /* device workspace for getrf */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void *h_work = nullptr;              /* host workspace for getrf */


    /* step 1: create cusolver handle, bind a stream */
    cudacheck( hipsolverDnCreate( &cusolverH ) );
    cudacheck( hipStreamCreate( &stream ) );
    cudacheck( hipsolverSetStream( cusolverH, stream ) );


    /* Create advanced params */
    hipsolverDnParams_t params;
    cudacheck( hipsolverDnCreateParams( &params ) );



    /* step 2: copy A to device */
    cudacheck(hipMalloc(&d_A, Neff*Neff*CD));
    cudacheck( hipMalloc( &d_Ipiv, sizeof( int64_t ) * Neff ) );
    cudacheck( hipMalloc( &d_info, sizeof( int ) ) );

    cudacheck(hipMemcpy(d_A, D_removed, Neff*Neff*CD, H2D));


    /* step 3: query working space of getrf */
    cudacheck( hipsolverDnXgetrf_bufferSize( cusolverH, NULL, Neff, Neff,
                                            HIP_C_64F, d_A, Neff, HIP_C_64F,
                                            &workspaceInBytesOnDevice, &workspaceInBytesOnHost ) );


    cudacheck(hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice));

    if (0 < workspaceInBytesOnHost) {
      h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
      if (h_work == nullptr) {
        throw std::runtime_error("Error: h_work not allocated.");
      }
    }


    /* step 4: LU factorization */
    cudacheck(hipsolverDnXgetrf(cusolverH, params, m, m, HIP_C_64F,
                               d_A, lda, d_Ipiv, HIP_C_64F, d_work,
                               workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info));

    cudacheck(hipMemcpy(&info, d_info, sizeof(int), D2H));
    cudacheck(hipMemcpy(LU, d_A, Neff*Neff*CD, D2H));

    std::printf("after Xgetrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    double log_det = 0.0;
    for(Idx i=0; i<Neff; i++) {
      assert( imag(LU[i*Neff+i])<1.0e-14 );
      log_det += std::log( real(LU[i*Neff+i]) );
    }
    std::cout << "log det = " << log_det << std::endl;




    // //============================================
    // // obsolete






    // // // Start timer
    // // hipEvent_t startEvent { nullptr };
    // // hipEvent_t stopEvent { nullptr };
    // // float       elapsed_gpu_ms {};
    // // cudacheck( hipEventCreate( &startEvent, hipEventBlockingSync ) );
    // // cudacheck( hipEventCreate( &stopEvent, hipEventBlockingSync ) );

    // std::printf( "Pivot is on : compute P*A = L*U\n" );

    // // std::memcpy(m_A, D_removed, Neff*Neff*CD);
    // // hipDeviceSynchronize( );
    // std::printf( "Using New Algo\n" );
    // cudacheck( hipsolverDnSetAdvOptions( params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0 ) );


    // /* Create advanced params */




    // // memory

    // void *bufferOnDevice { nullptr };
    // void *bufferOnHost { nullptr };


    // std::printf( "\nAllocate device workspace, lwork = %lu\n", workspaceInBytesOnDevice );
    // std::printf( "Allocate host workspace, lwork = %lu\n\n", workspaceInBytesOnHost );

    // // cudacheck( hipMallocManaged( &bufferOnDevice, workspaceInBytesOnDevice ) );
    // size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    // void *d_work = nullptr;              /* device workspace for getrf */
    // size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    // void *h_work = nullptr;              /* host workspace for getrf */

    // if ( 0 < workspaceInBytesOnHost ) {
    //   cudacheck( hipMallocManaged( &bufferOnHost, workspaceInBytesOnHost ) );
    //   assert( NULL != bufferOnHost );
    // }

    // // Create advanced params
    // // if ( algo == 0 ) {
    // //   std::printf( "Using New Algo\n" );
    // //   cudacheck( hipsolverDnSetAdvOptions( params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0 ) );
    // // } else {
    // //   std::printf( "Using Legacy Algo\n" );
    // //   cudacheck( hipsolverDnSetAdvOptions( params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_1 ) );
    // // }


    // /* step 4: LU factorization */
    // std::printf( "\nRunning GETRF\n" );
    // // cudacheck( hipEventRecord( startEvent ) );

    // cudacheck( hipsolverDnXgetrf( cusolverH,
    //                              params,
    //                              static_cast<int64_t>( Neff ),
    //                              static_cast<int64_t>( Neff ),
    //                              HIP_C_64F,
    //                              d_A,
    //                              static_cast<int64_t>( Neff ),
    //                              d_Ipiv,
    //                              HIP_C_64F,
    //                              bufferOnDevice,
    //                              workspaceInBytesOnDevice,
    //                              bufferOnHost,
    //                              workspaceInBytesOnHost,
    //                              d_info ) );

    // cudacheck(hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice));
    // if (0 < workspaceInBytesOnHost) {
    //   h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
    //   if (h_work == nullptr) {
    //     throw std::runtime_error("Error: h_work not allocated.");
    //   }
    // }

    // // Must be here to retrieve d_info
    // // cudacheck( hipStreamSynchronize( stream ) );
    // if ( *d_info ) {
    //   throw std::runtime_error( std::to_string( -*d_info ) + "-th parameter is wrong (hipsolverDnDgetrf) \n" );
    // }

    // // Stop timer
    // cudacheck( hipEventRecord( stopEvent ) );
    // cudacheck( hipEventSynchronize( stopEvent ) );

    // cudacheck( hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent ) );
    // double avg { elapsed_gpu_ms };
    // double flops { FLOPS_ZGETRF( Neff, Neff ) };
    // double perf { 1e-9 * flops / avg };
    // std::printf( "\nRuntime = %0.2f ms : @ %0.2f GFLOPs\n\n", avg, perf );



    // ===========================================



    // cudacheck( hipEventDestroy( startEvent ) );
    // cudacheck( hipEventDestroy( stopEvent ) );

    cudacheck( hipFree( d_A ) );
    cudacheck( hipFree( d_Ipiv ) );
    cudacheck( hipFree( d_info ) );
    cudacheck( hipFree( d_work ) );
    free(h_work);

    cudacheck(hipsolverDnDestroyParams(params));
    cudacheck( hipsolverDnDestroy( cusolverH ) );
    cudacheck( hipStreamDestroy( stream ) );

    // cudacheck( hipFree( bufferOnDevice ) );
    // cudacheck( hipFree( bufferOnHost ) );

  }


  free( e );
  free( D );
  free( D_removed );
  free( LU );
  hipDeviceReset();

  return 0;
}

